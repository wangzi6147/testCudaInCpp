#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void addWithCuda(int *c, int *b, int *a, int size){
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	addKernel << <1, size >> >(dev_c, dev_a, dev_b);
}
