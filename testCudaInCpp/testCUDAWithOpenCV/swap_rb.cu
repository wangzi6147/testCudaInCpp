#include "hip/hip_runtime.h"
#include <opencv2/core/cuda_devptrs.hpp>
#include "hip/hip_runtime.h"
#include ""

using namespace cv;
using namespace cv::gpu;
//�Զ����ں˺���
__global__ void swap_rb_kernel(const PtrStepSz<uchar3> src, PtrStep<uchar3> dst)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < src.cols && y < src.rows)
	{
		uchar3 v = src(y, x);
		dst(y, x) = make_uchar3(v.z, v.y, v.x);
	}
}

void swap_rb_caller(const PtrStepSz<uchar3>& src, PtrStep<uchar3> dst, hipStream_t stream)
{
	dim3 block(32, 8);
	dim3 grid((src.cols + block.x - 1) / block.x, (src.rows + block.y - 1) / block.y);

	swap_rb_kernel << <grid, block, 0, stream >> >(src, dst);
	if (stream == 0)
		hipDeviceSynchronize();
}