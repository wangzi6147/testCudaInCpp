#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void addWithCuda(int *c, int *b, int *a, int size){
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipSetDevice(0);
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
}
